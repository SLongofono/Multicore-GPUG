#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <fstream>
#include "ImageWriter.h"
#include <hip/hip_runtime.h>
#include "helpers.h"
#include <cmath>

#define DEBUG 1
#define DEVICE_NUM 0

using namespace std;


void __global__ kernelMaxImage(unsigned char *voxels, unsigned char *maxImage, float *weightedSums, float *globalMax, int nSheets){
	int myPos = blockIdx.x*blockDim.x + threadIdx.x;
	int localMax = 0;
	float norm = 1.0/nSheets;
	float weightedSum = 0.0;
	int imageSize = gridDim.x*blockDim.x;
	unsigned char curVal;

	for(int sh = 0; sh < nSheets; sh++){
		curVal = voxels[myPos + sh*imageSize];
		
		if(curVal > localMax){
			localMax = curVal;
		}
		weightedSum += norm * curVal * (sh+1);
		//weightedSum += norm * curVal*(nSheets - sh);
	}

	// Update output image for my pixel
	maxImage[myPos] = localMax;

	// Update weighted sums for my pixel
	weightedSums[myPos] = weightedSum;

	// Update global Max
	atomicMax(globalMax, weightedSum);
}


void __global__ kernelSumImage(float *weightedSums, unsigned char *sumImage, float *globalMax){
	int myPos = blockIdx.x*blockDim.x + threadIdx.x;
	float max = globalMax[0];
	float localMax = weightedSums[myPos];
	int result = (int)((localMax/max)*255.0);
	float diff = (localMax - max);
	if(diff < 0){
		diff *= -1.0;
	}
	//printf("Norm is %f, myVal is %f", norm, weightedSums[myPos]);
	// Formula : output = round( (p/globalMax)*255.0  )
	if((max - localMax) < 0.001){
		printf("MyPos: %d, globalMax: %f, localMax: %f, result: %d\n", myPos, max, localMax, result);
	}
	sumImage[myPos] = result;
}


int main(int argc, char **argv){
	
	/*
	 * Setup and housekeeping...
	 */
	if(argc < 7){
		cerr << "[ error ] Expected more arguments!" << endl;
		cerr << "Usage: ./project3 nRows nCols nSheets fileName projectionType outputFileNameBase" << endl;
		cerr << "Types: executable int int int string int string" << endl;
		cerr << "Exiting..." << endl;
		return -1;
	}

#if DEBUG
	// Dump device information
	dumpDevices();
#endif

	int nRows, nCols, nSheets, nVals, projType;
	std::string fileType(".png");
	nRows = atoi(argv[1]);
	nCols = atoi(argv[2]);
	nSheets = atoi(argv[3]);
	projType = atoi(argv[5]);
	nVals = nRows * nCols * nSheets;
	unsigned char *rawImageData = new unsigned char[nVals]();	
	unsigned char *d_voxels;
	unsigned char *d_maxImage, *h_maxImage;
	unsigned char *d_sumImage, *h_sumImage;
	int resultSize;
	float *d_weightedSums;
	float *d_globalMax;

	
	ifstream infile(argv[4]);
	if(!infile.good()){
		cerr << "[ error ] Bad input filename.  Exiting..." << endl;
		return -1;
	}
	infile.read( reinterpret_cast<char *>(rawImageData), nVals);
	infile.close();


	/*
	 * Copy voxel data to the GPU
	 *
	 * We can assume that device 0 is valid if it exists.  Here, I
	 * explicitly set it to use the device I want, the Quadro 6000.  Not
	 * necessary, but I wanted to leave this here for future reference.
	 */
	hipSetDevice(DEVICE_NUM);
	validate(hipMalloc((void **)&d_voxels, nVals*sizeof(unsigned char)));

	/*
	 * Configure projection-specific details and launch kernels.  Rather
	 * than rely on the GPU to try and traverse differently based on the
	 * projection, we swap out the data in-place to the desired
	 * projection, allowing us to optimize memory access in terms of the
	 * resulting 2D image dimensions.
	 */
	
	// Re-flatten array per projection
	projection(rawImageData, nRows, nCols, nSheets, projType);
	validate(hipMemcpy(d_voxels, rawImageData, nVals*sizeof(unsigned char),hipMemcpyHostToDevice));

	// Issue kernels
	switch(projType){
		case 1:	// Note: need braces to restrict scope of the local variables
			{
				cout << "Projection type " << projType << endl;
				resultSize = nCols*nRows*sizeof(unsigned char);
				h_maxImage = new unsigned char[nCols*nRows];
				h_sumImage = new unsigned char[nCols*nRows];
				validate(hipMalloc((void **)&d_maxImage, resultSize));
				validate(hipMalloc((void **)&d_sumImage, resultSize));
				validate(hipMalloc((void **)&d_weightedSums, nCols*nRows*sizeof(float)));
				validate(hipMalloc((void **)&d_globalMax, sizeof(float)));
			
				kernelMaxImage<<<nCols, nRows>>>(d_voxels,d_maxImage, d_weightedSums, d_globalMax, nSheets);
				validate(hipPeekAtLastError()); // Check invalid launch
				validate(hipDeviceSynchronize()); // Check runtime error

				kernelSumImage<<<nCols, nRows>>>(d_weightedSums, d_sumImage, d_globalMax);
				validate(hipPeekAtLastError());
				validate(hipDeviceSynchronize());
			}
			break;
		case 2:
			cout << "Projection type " << projType << endl;
			resultSize = nCols*nRows*sizeof(unsigned char);
			h_maxImage = new unsigned char[nCols*nRows];
			hipMalloc((void **)&d_maxImage, resultSize);
			hipMalloc((void **)&d_sumImage, resultSize);
			hipMalloc((void **)&d_weightedSums, nCols*nRows*sizeof(float));

			break;
		case 3:
			cout << "Projection type " << projType << endl;
			resultSize = nSheets*nRows*sizeof(unsigned char);
			h_maxImage = new unsigned char[nSheets*nRows];
			hipMalloc((void **)&d_maxImage, resultSize);
			hipMalloc((void **)&d_sumImage, resultSize);
			hipMalloc((void **)&d_weightedSums, nSheets*nRows*sizeof(float));

			break;
		case 4:
			cout << "Projection type " << projType << endl;
			resultSize = nSheets*nRows*sizeof(unsigned char);
			h_maxImage = new unsigned char[nSheets*nRows];
			hipMalloc((void **)&d_maxImage, resultSize);
			hipMalloc((void **)&d_sumImage, resultSize);
			hipMalloc((void **)&d_weightedSums, nSheets*nRows*sizeof(float));

			break;
		case 5:
			cout << "Projection type " << projType << endl;
			resultSize = nCols*nSheets*sizeof(unsigned char);
			h_maxImage = new unsigned char[nCols*nSheets];
			hipMalloc((void **)&d_maxImage, resultSize);
			hipMalloc((void **)&d_sumImage, resultSize);
			hipMalloc((void **)&d_weightedSums, nCols*nSheets*sizeof(float));

			break;
		case 6:
			cout << "Projection type " << projType << endl;
			resultSize = nCols*nSheets*sizeof(unsigned char);
			h_maxImage = new unsigned char[nCols*nSheets];
			hipMalloc((void **)&d_maxImage, resultSize);
			hipMalloc((void **)&d_sumImage, resultSize);
			hipMalloc((void **)&d_weightedSums, nCols*nSheets*sizeof(float));

			break;
		default:
			cerr << "[ error ] '" << projType << "' is not a valid projection type, please select from [1,6]" << endl;
			delete [] rawImageData;
			return -1;
	}

	/*
	 * Retrieve results
	 */
	validate(hipMemcpy(h_maxImage, d_maxImage, resultSize, hipMemcpyDeviceToHost)); 
	validate(hipMemcpy(h_sumImage, d_sumImage, resultSize, hipMemcpyDeviceToHost)); 

	/*
	 * Write results
	 */
	writeImage(argv[6] + std::string("_max.png"), h_maxImage, projType, nRows, nCols, nSheets);
	writeImage(argv[6] + std::string("_sum.png"), h_sumImage, projType, nRows, nCols, nSheets);

	/*
	 * Clean up
	 */
	hipFree(d_maxImage);
	hipFree(d_sumImage);
	hipFree(d_weightedSums);
	hipFree(d_globalMax);

	delete [] rawImageData;
	delete [] h_maxImage;
	delete [] h_sumImage;

	return 0;
}
